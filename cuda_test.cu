#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>

std::string detect_and_print() {
    // Create an ostringstream object
    std::ostringstream oss;
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        oss << "Error: " << hipGetErrorString(error_id) << std::endl;
        return oss.str();
    }

    if (deviceCount == 0) {
        oss << "No CUDA-capable devices detected." << std::endl;
        return oss.str();
    }

    oss << "Number of CUDA-capable devices: " << deviceCount << std::endl;

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        oss << "\nDevice " << device << ": " << deviceProp.name << std::endl;
        oss << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        oss << "  Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        oss << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        oss << "  Maximum Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        oss << "  Maximum Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        oss << "  Maximum Grid Size: [" 
                  << deviceProp.maxGridSize[0] << ", " 
                  << deviceProp.maxGridSize[1] << ", " 
                  << deviceProp.maxGridSize[2] << "]" << std::endl;
        oss << "  Maximum Threads Dimensions: [" 
                  << deviceProp.maxThreadsDim[0] << ", " 
                  << deviceProp.maxThreadsDim[1] << ", " 
                  << deviceProp.maxThreadsDim[2] << "]" << std::endl;
    }

    return oss.str();
}
